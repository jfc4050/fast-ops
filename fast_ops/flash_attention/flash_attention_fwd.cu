#include "hip/hip_runtime.h"
#include "ATen/ops/zeros_like.h"
#include "c10/core/ScalarType.h"
#include <ATen/core/TensorAccessor.h>
#include <cute/tensor.hpp>
#include <cutlass/cutlass.h>
#include <torch/extension.h>

#include "common/launch_utils.h"
#include "cute/layout.hpp"
#include "cute/pointer.hpp"
#include "cute/swizzle_ptr.hpp"

template <typename scalar_t_pt, int BLOCK_M, int BLOCK_N, int BLOCK_D>
__global__ void flash_attn_fwd_kernel(
    const at::PackedTensorAccessor32<scalar_t_pt, 4, at::RestrictPtrTraits>
        Q_accessor,
    const at::PackedTensorAccessor32<scalar_t_pt, 4, at::RestrictPtrTraits>
        K_accessor,
    const at::PackedTensorAccessor32<scalar_t_pt, 4, at::RestrictPtrTraits>
        V_accessor,
    at::PackedTensorAccessor32<scalar_t_pt, 4, at::RestrictPtrTraits>
        O_accessor) {

  // map PyTorch type to CUTLASS type
  using scalar_t = typename cutlass_t<scalar_t_pt>::value;

  __shared__ scalar_t Qi_smem[BLOCK_M * BLOCK_D];

  const int seqlen_m = Q_accessor.size(3); // number of queries
  const int seqlen_n = K_accessor.size(3); // number of keys/values

  const int batch_idx = blockIdx.x;
  const int head_idx = blockIdx.y;
  const int seq_chunk_m_idx = blockIdx.z;
  const int start_m = seq_chunk_m_idx * BLOCK_M;

  // represent full tensors
  auto Q = cute::make_tensor(
      cute::make_gmem_ptr(
          reinterpret_cast<scalar_t *>(Q_accessor[batch_idx][head_idx].data())),
      cute::make_shape(Q_accessor.size(3), Q_accessor.size(4)));
  // TODO. do K and V as well

  // represent SRAM tiles
  // TODO. double check stride
  auto Qi = cute::make_tensor(cute::make_smem_ptr(Qi_smem),
                              cute::make_shape(BLOCK_M, BLOCK_D), BLOCK_M);

  for (int seq_chunk_n_start = 0; seq_chunk_n_start < seqlen_n;
       seq_chunk_n_start += BLOCK_N) {
    ;
  }
}

at::Tensor flash_attn_fwd_cuda(at::Tensor Q, at::Tensor K, at::Tensor V) {
  const int head_dim = Q.size(-1);
  const float sm_scale = 1.0 / sqrt(head_dim);

  at::Tensor O = torch::zeros_like(Q);

  const int batch_sz = O.size(0);
  const int n_heads = O.size(1);
  const int seqlen_m = O.size(2);

  constexpr int BLOCK_M = 128;
  constexpr int BLOCK_N = 128;
  constexpr int BLOCK_D = 128; // TODO. dispatch based on runtime headdim
  const int n_blocks_m = (seqlen_m + BLOCK_M - 1) / BLOCK_M;

  AT_DISPATCH_HALF_TYPES(
      Q.scalar_type(), "flash_attn_fwd", ([&] {
        // TODO. double check block dim
        flash_attn_fwd_kernel<scalar_t, BLOCK_M, BLOCK_N, BLOCK_D>
            <<<dim3(batch_sz, n_heads, n_blocks_m), dim3(BLOCK_M, BLOCK_N)>>>(
                Q.packed_accessor32<scalar_t, 4, at::RestrictPtrTraits>(),
                K.packed_accessor32<scalar_t, 4, at::RestrictPtrTraits>(),
                V.packed_accessor32<scalar_t, 4, at::RestrictPtrTraits>(),
                O.packed_accessor32<scalar_t, 4, at::RestrictPtrTraits>());
      }));

  return O;
}
